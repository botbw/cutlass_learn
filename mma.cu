#include "hip/hip_runtime.h"
// CUTLASS_REPO_PATH=~/cutlass nvcc -arch=sm_80 -I${CUTLASS_REPO_PATH}/include mma.cu && ./a.out && rm a.out
#include <cstdlib>
#include <stdio.h>
#include <iostream>

#include <hip/hip_runtime.h>

#include "cute/tensor.hpp"
#include "cute/arch/mma_sm80.hpp"
#include "utils.cuh"

__global__ void trivialGemm(double *A, double *B, double *C, int M, int N, int K)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < M && j < N)
    {
        double sum = 0.0f;
        for (int k = 0; k < K; k++)
        {
            sum += A[i + k * M] * B[k + j * K];
        }
        C[i + j * M] = sum; // Store result in C
    }
}

constexpr int BM = 16;
constexpr int BN = 16;
constexpr int BK = 8;

template <int BM, int BN, int BK>
__global__ void trivialMMAWithoutCuTe(const double *A, const double *B, double *C, int m, int n, int k)
{
    /*
        A, B, C are column-major matrices
        A: (m, k):(1, m)
        B: (k, n):(1, k)
        C: (m, n):(1, m)
    */
    // sanity check
    assert(m % BM == 0);
    assert(n % BN == 0);
    assert(k % BK == 0);
    static_assert(BM % 8 == 0);
    static_assert(BK % 4 == 0);
    static_assert(BN % 4 == 0);

    constexpr int WARP_REP_M = BM / 8;
    // constexpr int WARP_REP_N = BN / 8;
    constexpr int WARP_SLICE_K = BK / 4;

    int warpId = threadIdx.x / 32;
    int warp_i = warpId % WARP_REP_M;
    int warp_j = warpId / WARP_REP_M;
    int lane_id = threadIdx.x % 32;

    double c0_acc, c1_acc;
    c0_acc = c1_acc = 0;
    for (int blk_k = 0; blk_k < k / BK; blk_k++)
    {
        double a0, b0, d0, d1;
#pragma unroll
        for (int warp_k = 0; warp_k < WARP_SLICE_K; warp_k++)
        { // each tensor core takes k = 4
            // from nvidia official guide
            int a_row = lane_id / 4;
            int a_col = lane_id % 4;
            // A is 8 x 4 per warp
            a0 = // A[blockIdx.x * BM + warp_i * 8 + a_row , blk_k * BK + warp_k * 4 + warp_j * 4 + a_col]
                *(A + blockIdx.x * BM + warp_i * 8 + a_row + (blk_k * BK + warp_k * 4 + a_col) * m);

            // from nvidia official guide
            int b_row = lane_id % 4;
            int b_col = lane_id / 4;
            // B is 4 x 8 per warp
            b0 = // B[blk_k * BK + warp_k * 4 + warp_i * 4 + b_row, blockIdx.y * BN + warp_j * 8 + b_col]
                *(B + blk_k * BK + warp_k * 4 + b_row + (blockIdx.y * BN + warp_j * 8 + b_col) * k);

            // wait for warp to load value to register and mma
            cute::SM80_8x8x4_F64F64F64F64_TN::fma(d0, d1, a0, b0, 0, 0);

            // accumulate
            c0_acc += d0;
            c1_acc += d1;
        }
    }
    // from nvidia official guide
    int c_groupID = lane_id / 4;
    int c_threadID_in_group = lane_id % 4;
    int c_row = c_groupID;
    int c0_col = c_threadID_in_group * 2;
    int c1_col = c_threadID_in_group * 2 + 1;

    // C is 8 x 8 per warp
    double *pC0 = // &C[blockIdx.x * BM + warp_i * 8 + c_row, blockIdx.y * BN + warp_j * 8 + c0_col]
        C + blockIdx.x * BM + warp_i * 8 + c_row + (blockIdx.y * BN + warp_j * 8 + c0_col) * m;
    double *pC1 = // &C[blockIdx.x * BM + warp_i * 8 + c_row, blockIdx.y * BN + warp_j * 8 + c1_col]
        C + blockIdx.x * BM + warp_i * 8 + c_row + (blockIdx.y * BN + warp_j * 8 + c1_col) * m;

    // save result
    *pC0 = c0_acc; // c0_acc;
    *pC1 = c1_acc; // c1_acc;
}

template <int BM, int BN, int BK>
__global__ void trivialMMAWithCuTe(const double *pA, const double *pB, double *pC, int m, int n, int k)
{
    /*
        A, B, C are column-major matrices
        A: (m, k):(1, m)
        B: (k, n):(1, k)
        C: (m, n):(1, m)
    */
    assert(m % BM == 0);
    assert(n % BN == 0);
    assert(k % BK == 0);
    static_assert(BM % 8 == 0);
    static_assert(BK % 4 == 0);
    static_assert(BN % 4 == 0);

    constexpr int WARP_REP_M = BM / 8;
    // constexpr int WARP_REP_N = BN / 8;
    constexpr int WARP_SLICE_K = BK / 4;

    using namespace cute;
    Tensor A = make_tensor(pA, make_shape(m, k), make_stride(1, m));
    Tensor B = make_tensor(pB, make_shape(k, n), make_stride(1, k));
    Tensor C = make_tensor(pC, make_shape(m, n), make_stride(1, m));

    // tile abc and get cooresponding subtensor
    Tensor gA = zipped_divide(A, make_shape(BM, BK))(_, make_coord(blockIdx.x, _)); // (m, k) -> ((BM, BK), (m / BM, k / BK)) -> ((BM, BK), k / BK)
    Tensor gB = zipped_divide(B, make_shape(BK, BN))(_, make_coord(_, blockIdx.y));
    Tensor gC = zipped_divide(C, make_shape(BM, BN))(make_coord(_, _), make_coord(blockIdx.x, blockIdx.y));

    int warpId = threadIdx.x / 32;
    int warp_i = warpId % WARP_REP_M;
    int warp_j = warpId / WARP_REP_M;
    int lane_id = threadIdx.x % 32;

    double c0_acc, c1_acc;
    c0_acc = c1_acc = 0;
    for (int blk_k = 0; blk_k < k / BK; blk_k++)
    {
        double a0, b0, d0, d1;
        Tensor blkA = gA(make_coord(_, _), blk_k); // ((BM, BK), k / BK) -> (BM, BK)
        Tensor blkB = gB(make_coord(_, _), blk_k);
        Tensor blkA_sliced_k = zipped_divide(blkA, Shape<_8, _4>{}); // (BM, BK) -> ((8, 4), (WARP_REP_M, WARP_SLICE_K))
        Tensor blkB_sliced_k = zipped_divide(blkB, Shape<_4, _8>{}); // (BK, BN) -> ((4, 8), (WARP_SLICE_K, WARP_REP_N))
#pragma unroll
        for (int warp_k = 0; warp_k < WARP_SLICE_K; warp_k++)
        {
            auto ALayout = MMA_Traits<SM80_8x8x4_F64F64F64F64_TN>::ALayout{};
            Tensor warp_a = blkA_sliced_k(make_coord(_, _), make_coord(warp_i, warp_k)); // ((8, 4), (WARP_REP_M, WARP_SLICE_K)) -> (8, 4)
            a0 = warp_a(ALayout(lane_id, 0));

            // Note: B is (n, k) in CuTe, but (k, n) in this code
            auto BLayout = MMA_Traits<SM80_8x8x4_F64F64F64F64_TN>::BLayout{};
            Tensor warp_b = blkB_sliced_k(make_coord(_, _), make_coord(warp_k, warp_j));
            int b_nk = BLayout(lane_id, 0);
            int b_i = b_nk / 8, b_j = b_nk % 8;
            b0 = warp_b(b_i, b_j);

            cute::SM80_8x8x4_F64F64F64F64_TN::fma(d0, d1, a0, b0, 0, 0);

            c0_acc += d0;
            c1_acc += d1;
        }
    }
    Tensor warp_c = zipped_divide(gC, Shape<_8, _8>{})(make_coord(_, _), make_coord(warp_i, warp_j)); // ((BM, BN), (WARP_REP_M, WARP_REP_N)) -> ((8, 8), (WARP_REP_M, WARP_REP_N))
    auto CLayout = MMA_Traits<SM80_8x8x4_F64F64F64F64_TN>::CLayout{};

    warp_c(CLayout(lane_id, 0)) = c0_acc;
    warp_c(CLayout(lane_id, 1)) = c1_acc;
}

template <int BM, int BN, int BK, typename TiledMMA>
__global__ void cuteStyleGemm(double *pA, double *pB, double *pC, int m, int n, int k, TiledMMA tiled_mma)
{
    /*
        A, B, C are column-major matrices
        A: (m, k):(1, m)
        B: (k, n):(1, k)
        C: (m, n):(1, m)
    */
    assert(m % BM == 0);
    assert(n % BN == 0);
    assert(k % BK == 0);
    static_assert(BM % 8 == 0);
    static_assert(BK % 4 == 0);
    static_assert(BN % 4 == 0);

    using namespace cute;
    Tensor A = make_tensor(pA, make_shape(m, k), make_stride(1, m));
    Tensor B = make_tensor(pB, make_shape(n, k), make_stride(k, 1)); // B is (k, n) in CuTe, but (n, k) in this code
    Tensor C = make_tensor(pC, make_shape(m, n), make_stride(1, m));

    Tensor gA = local_tile(A, Shape<Int<BM>, Int<BK>>{}, make_coord(blockIdx.x, _));          // same as zipped_divide(A, make_shape(BM, BK))(_, make_coord(blockIdx.x, _));
    Tensor gB = local_tile(B, Shape<Int<BN>, Int<BK>>{}, make_coord(blockIdx.y, _));          // zipped_divide(B, make_shape(BK, BN))(_, make_coord(_, blockIdx.y));
    Tensor gC = local_tile(C, Shape<Int<BM>, Int<BN>>{}, make_coord(blockIdx.x, blockIdx.y)); // zipped_divide(C, make_shape(BM, BN))(make_coord(_, _), make_coord(blockIdx.x, blockIdx.y));

    auto thr_mma = tiled_mma.get_slice(threadIdx.x);

    auto tAgA = thr_mma.partition_A(gA);
    auto tBgB = thr_mma.partition_B(gB);
    auto tCgC = thr_mma.partition_C(gC);

    auto tArA = thr_mma.partition_fragment_A(gA(_, _, 0));
    auto tBrB = thr_mma.partition_fragment_B(gB(_, _, 0));
    auto tCrC = thr_mma.partition_fragment_C(gC(_, _));

    clear(tCrC);
#pragma unroll
    for (int blk_k = 0; blk_k < k / BK; blk_k++)
    {
        copy(tAgA(_, _, _, blk_k), tArA);
        copy(tBgB(_, _, _, blk_k), tBrB);
        cute::gemm(tiled_mma, tCrC, tArA, tBrB, tCrC);
    }
    copy(tCrC, tCgC);
}


__global__ void checkArrayEquality(double* array1, double* array2, bool* result, int length) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < length) {
        if (array1[idx] != array2[idx]) {
            atomicExch((int*)result, int(false));
        }
    }
}

void assertEqual(double* A, double* B, int size) {
    bool h_result = true;
    bool* d_result;
    double *d_A, *d_B;

    hipMalloc((void**)&d_result, sizeof(bool));
    hipMalloc((void**)&d_A, size * sizeof(double));
    hipMalloc((void**)&d_B, size * sizeof(double));
    hipMemcpy(d_result, &h_result, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_A, A, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    checkArrayEquality<<<numBlocks, blockSize>>>(A, B, d_result, size);
    hipDeviceSynchronize();
    hipMemcpy(&h_result, d_result, sizeof(bool), hipMemcpyDeviceToHost);

    hipFree(d_result);
    hipFree(d_A);
    hipFree(d_B);

    assert(h_result);
}

void printMatrix(double *A, int m, int n)
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            // A[i, j]
            std::cout << A[i + j * m] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl
              << std::endl;
}

int main(int argc, char const *argv[])
{
    int m = 2048;
    int n = 2048;
    int k = 2048;
    // parse args m n k if provided
    if (argc == 4)
    {
        m = std::atoi(argv[1]);
        n = std::atoi(argv[2]);
        k = std::atoi(argv[3]);
    }
    double *A, *B, *C, *C_ref;
    hipHostAlloc(reinterpret_cast<void **>(&A), m * k * sizeof(double), hipHostMallocDefault);
    hipHostAlloc(reinterpret_cast<void **>(&B), k * n * sizeof(double), hipHostMallocDefault);
    hipHostAlloc(reinterpret_cast<void **>(&C), m * n * sizeof(double), hipHostMallocDefault);
    hipHostAlloc(reinterpret_cast<void **>(&C_ref), m * n * sizeof(double), hipHostMallocDefault);
    randn(A, m * k, 0, 10);
    randn(B, k * n, 0, 10);

    double *dA, *dB, *dC;
    hipMalloc(reinterpret_cast<void **>(&dA), m * k * sizeof(double));
    hipMalloc(reinterpret_cast<void **>(&dB), k * n * sizeof(double));
    hipMalloc(reinterpret_cast<void **>(&dC), m * n * sizeof(double));
    hipMemcpy(dA, A, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, k * n * sizeof(double), hipMemcpyHostToDevice);

    {
        // test trivialGemm
        randn(C, m * n, 0, 100);
        dim3 threads(16, 16);
        dim3 blocks(m / 16, n / 16);
        time_t start, end;
        start = clock();
        trivialGemm<<<blocks, threads>>>(dA, dB, dC, m, n, k);
        hipError_t err = hipDeviceSynchronize();
        end = clock();
        printf("CUDA error: %s\n", hipGetErrorString(err));
        printf("Runtime of trivialGemm: %f\n", (double)(end - start) / CLOCKS_PER_SEC);
        hipMemcpy(C_ref, dC, m * n * sizeof(double), hipMemcpyDeviceToHost);
    }

    {
        // test trivialMMAWithoutCuTe
        randn(C, m * n, 0, 100);
        dim3 threads(BM * BN / 2);
        dim3 blocks(m / BM, n / BN);
        time_t start, end;
        start = clock();
        trivialMMAWithoutCuTe<BM, BN, BK><<<blocks, threads>>>(dA, dB, dC, m, n, k);
        hipError_t err = hipDeviceSynchronize();
        printf("CUDA error: %s\n", hipGetErrorString(err));
        end = clock();
        printf("Runtime of trivialMMAWithoutCuTe: %f\n", (double)(end - start) / CLOCKS_PER_SEC);
        hipMemcpy(C, dC, m * n * sizeof(double), hipMemcpyDeviceToHost);
        assertEqual(C, C_ref, m * n);
        printf("trivialMMAWithoutCuTe passed\n");
    }

    {
        // test trivialMMAWithCuTe
        randn(C, m * n, 0, 100);
        dim3 threads(BM * BN / 2);
        dim3 blocks(m / BM, n / BN);
        time_t start, end;
        start = clock();
        trivialMMAWithCuTe<BM, BN, BK><<<blocks, threads>>>(dA, dB, dC, m, n, k);
        hipError_t err = hipDeviceSynchronize();
        end = clock();
        printf("CUDA error: %s\n", hipGetErrorString(err));
        printf("Runtime of trivialMMAWithCuTe: %f\n", (double)(end - start) / CLOCKS_PER_SEC);
        hipMemcpy(C, dC, m * n * sizeof(double), hipMemcpyDeviceToHost);
        assertEqual(C, C_ref, m * n);
        printf("trivialMMAWithCuTe passed\n");
    }

    {
        // test cuteStyleGemm
        using namespace cute;
        randn(C, m * n, 0, 100);
        auto tiled_mma = cute::make_tiled_mma(SM80_8x8x4_F64F64F64F64_TN{}, make_layout(Shape<Int<BM / 8>, Int<BN / 8>>{}));
        dim3 threads(size(tiled_mma));
        dim3 blocks(m / BM, n / BN);
        time_t start, end;
        start = clock();
        cuteStyleGemm<BM, BN, BK><<<blocks, threads>>>(dA, dB, dC, m, n, k, tiled_mma);
        hipError_t err = hipDeviceSynchronize();
        end = clock();
        printf("CUDA error: %s\n", hipGetErrorString(err));
        printf("Runtime of cuteStyleGemm: %f\n", (double)(end - start) / CLOCKS_PER_SEC);
        hipMemcpy(C, dC, m * n * sizeof(double), hipMemcpyDeviceToHost);
        assertEqual(C, C_ref, m * n);
        printf("cuteStyleGemm passed\n");
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);
    hipHostFree(C_ref);
    return 0;
}