#include "hip/hip_runtime.h"
// nvcc -arch=sm_80 -I${CUTLASS_REPO_PATH}/include mma.cu && ./a.out && rm a.out
#include <cstdlib>
#include <stdio.h>
#include <iostream>

#include "cute/tensor.hpp"
#include "cute/arch/mma_sm80.hpp"
#include "utils.cuh"

__global__ void trivialGemm(double *A, double *B, double *C, int M, int N, int K) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int j = blockIdx.y * blockDim.y + threadIdx.y; 

    if (i < M && j < N) {
        double sum = 0.0f;
        for (int k = 0; k < K; k++) {
            sum += A[i + k * M] * B[k + j * K];
        }
        C[i + j * M] = sum;  // Store result in C
    }
}

constexpr int BM = 16;
constexpr int BN = 16;
constexpr int BK = 8;

template <int BM, int BN, int BK>
__global__ void trivialMMAWithoutCuTe(const double *A, const double *B, double *C, int m, int n, int k) {
    /*
        A, B, C are column-major matrices
        A: (m, k):(1, m)
        B: (k, n):(1, k)
        C: (m, n):(1, m)
    */
    // sanity check
    assert(m % BM == 0);
    assert(n % BN == 0);
    assert(k % BK == 0);
    static_assert(BM % 8 == 0);
    static_assert(BK % 4 == 0);
    static_assert(BN % 4 == 0);

    constexpr int WARP_REP_M = BM / 8;
    // constexpr int WARP_REP_N = BN / 8;
    constexpr int WARP_SLICE_K = BK / 4;

    int warpId = threadIdx.x / 32;
    int warp_i = warpId % WARP_REP_M;
    int warp_j = warpId / WARP_REP_M;
    int lane_id = threadIdx.x % 32;

    double c0_acc, c1_acc;
    c0_acc = c1_acc = 0;
    for (int blk_k = 0; blk_k < k / BK; blk_k++) {
        double a0, b0, d0, d1;
        #pragma unroll
        for (int warp_k = 0; warp_k < WARP_SLICE_K; warp_k++) {  // each tensor core takes k = 4
            // from nvidia official guide
            int a_row = lane_id / 4;
            int a_col = lane_id % 4;
            // A is 8 x 4 per warp
            a0 = // A[blockIdx.x * BM + warp_i * 8 + a_row , blk_k * BK + warp_k * 4 + warp_j * 4 + a_col]
                *(A + blockIdx.x * BM + warp_i * 8 + a_row + (blk_k * BK + warp_k * 4 + a_col) * m);

            // from nvidia official guide
            int b_row = lane_id % 4;
            int b_col = lane_id / 4;
            // B is 4 x 8 per warp
            b0 = // B[blk_k * BK + warp_k * 4 + warp_i * 4 + b_row, blockIdx.y * BN + warp_j * 8 + b_col]
                *(B + blk_k * BK + warp_k * 4 + b_row + (blockIdx.y * BN + warp_j * 8 + b_col) * k);

            // wait for warp to load value to register and mma
            cute::SM80_8x8x4_F64F64F64F64_TN::fma(d0, d1, a0, b0, 0, 0);

            // accumulate
            c0_acc += d0;
            c1_acc += d1;
        }
    }
    // from nvidia official guide
    int c_groupID = lane_id / 4;
    int c_threadID_in_group = lane_id % 4;
    int c_row = c_groupID;
    int c0_col = c_threadID_in_group * 2;
    int c1_col = c_threadID_in_group * 2 + 1;

    // C is 8 x 8 per warp
    double *pC0 = // &C[blockIdx.x * BM + warp_i * 8 + c_row, blockIdx.y * BN + warp_j * 8 + c0_col]
        C + blockIdx.x * BM + warp_i * 8 + c_row + (blockIdx.y * BN + warp_j * 8 + c0_col) * m;
    double *pC1 = // &C[blockIdx.x * BM + warp_i * 8 + c_row, blockIdx.y * BN + warp_j * 8 + c1_col]
        C + blockIdx.x * BM + warp_i * 8 + c_row + (blockIdx.y * BN + warp_j * 8 + c1_col) * m;

    // save result
    *pC0 = c0_acc; //c0_acc;
    *pC1 = c1_acc; //c1_acc;
}


template <int BM, int BN, int BK>
__global__ void trivialMMAWithCuTe(const double *pA, const double *pB, double *pC, int m, int n, int k) {
    /*
        A, B, C are column-major matrices
        A: (m, k):(1, m)
        B: (k, n):(1, k)
        C: (m, n):(1, m)
    */
    assert(m % BM == 0);
    assert(n % BN == 0);
    assert(k % BK == 0);
    static_assert(BM % 8 == 0);
    static_assert(BK % 4 == 0);
    static_assert(BN % 4 == 0);

    constexpr int WARP_REP_M = BM / 8;
    // constexpr int WARP_REP_N = BN / 8;
    constexpr int WARP_SLICE_K = BK / 4;

    using namespace cute;
    Tensor A = make_tensor(pA, make_shape(m, k), make_stride(1, m));
    Tensor B = make_tensor(pB, make_shape(k, n), make_stride(1, k));
    Tensor C = make_tensor(pC, make_shape(m, n), make_stride(1, m));

    // tile abc and get cooresponding subtensor
    Tensor gA = zipped_divide(A, make_shape(BM, BK))(_, make_coord(blockIdx.x, _)); // (m, k) -> ((BM, BK), (m / BM, k / BK)) -> ((BM, BK), k / BK)
    Tensor gB = zipped_divide(B, make_shape(BK, BN))(_, make_coord(_, blockIdx.y));
    Tensor gC = zipped_divide(C, make_shape(BM, BN))(make_coord(_, _), make_coord(blockIdx.x, blockIdx.y));

    int warpId = threadIdx.x / 32;
    int warp_i = warpId % WARP_REP_M;
    int warp_j = warpId / WARP_REP_M;
    int lane_id = threadIdx.x % 32;

    double c0_acc, c1_acc;
    c0_acc = c1_acc = 0;
    for (int blk_k = 0; blk_k < k / BK; blk_k++) {
        double a0, b0, d0, d1;
        Tensor blkA = gA(make_coord(_, _), blk_k); // ((BM, BK), k / BK) -> (BM, BK)
        Tensor blkB = gB(make_coord(_, _), blk_k);
        Tensor blkA_sliced_k = zipped_divide(blkA, Shape<_8, _4>{});  // (BM, BK) -> ((8, 4), (WARP_REP_M, WARP_SLICE_K))
        Tensor blkB_sliced_k = zipped_divide(blkB, Shape<_4, _8>{});  // (BK, BN) -> ((4, 8), (WARP_SLICE_K, WARP_REP_N))
        #pragma unroll
        for (int warp_k = 0; warp_k < WARP_SLICE_K; warp_k++) {
            auto ALayout = MMA_Traits<SM80_8x8x4_F64F64F64F64_TN>::ALayout{};
            Tensor warp_a = blkA_sliced_k(make_coord(_, _), make_coord(warp_i, warp_k));  // ((8, 4), (WARP_REP_M, WARP_SLICE_K)) -> (8, 4)
            a0 = warp_a(ALayout(lane_id, 0));

            // Note: B is (n, k) in CuTe, but (k, n) in this code
            auto BLayout = MMA_Traits<SM80_8x8x4_F64F64F64F64_TN>::BLayout{};
            Tensor warp_b = blkB_sliced_k(make_coord(_, _), make_coord(warp_k, warp_j));
            int b_nk = BLayout(lane_id, 0);
            int b_i = b_nk / 8, b_j = b_nk % 8;
            b0 = warp_b(b_i, b_j);

            cute::SM80_8x8x4_F64F64F64F64_TN::fma(d0, d1, a0, b0, 0, 0);

            c0_acc += d0;
            c1_acc += d1;
        }
    }
    Tensor warp_c = zipped_divide(gC, Shape<_8, _8>{})(make_coord(_, _), make_coord(warp_i, warp_j)); // ((BM, BN), (WARP_REP_M, WARP_REP_N)) -> ((8, 8), (WARP_REP_M, WARP_REP_N))
    auto CLayout = MMA_Traits<SM80_8x8x4_F64F64F64F64_TN>::CLayout{};

    warp_c(CLayout(lane_id, 0)) = c0_acc;
    warp_c(CLayout(lane_id, 1)) = c1_acc;
}

void refGemm(double *A, double *B, double *C, int m, int n, int k) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            double sum = 0;
            for (int l = 0; l < k; l++) {
                // A[i, l] * B[l, j]
                sum += A[i + l * m] * B[l + j * k];
            }
            // C[i, j]
            C[i + j * m] = sum;
        }
    }
}

void assertEqual(double *A, double *B, int size) {
    for (int i = 0; i < size; i++) {
        assert(A[i] == B[i]);
    }
}

void printMatrix(double *A, int m, int n) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            // A[i, j]
            std::cout << A[i + j * m] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl << std::endl;
}

int main(int argc, char const *argv[])
{
    int m = 2048;
    int n = 2048;
    int k = 2048;
     // parse args m n k if provided
    if (argc == 4) {
        m = std::atoi(argv[1]);
        n = std::atoi(argv[2]);
        k = std::atoi(argv[3]);
    }
    double *A, *B, *C, *C_ref;
    hipHostAlloc(reinterpret_cast<void**>(&A), m * k * sizeof(double), hipHostMallocDefault);
    hipHostAlloc(reinterpret_cast<void**>(&B), k * n * sizeof(double), hipHostMallocDefault);
    hipHostAlloc(reinterpret_cast<void**>(&C), m * n * sizeof(double), hipHostMallocDefault);
    hipHostAlloc(reinterpret_cast<void**>(&C_ref), m * n * sizeof(double), hipHostMallocDefault);
    randn(A, m * k, 0, 10);
    randn(B, k * n, 0, 10);
    refGemm(A, B, C_ref, m, n, k);

    double *dA, *dB, *dC;
    hipMalloc(reinterpret_cast<void**>(&dA), m * k * sizeof(double));
    hipMalloc(reinterpret_cast<void**>(&dB), k * n * sizeof(double));
    hipMalloc(reinterpret_cast<void**>(&dC), m * n * sizeof(double));
    hipMemcpy(dA, A, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, k * n * sizeof(double), hipMemcpyHostToDevice);

    {
        // test trivialGemm
        randn(C, m * n, 0, 100);
        dim3 threads(16, 16);
        dim3 blocks(m / 16, n / 16);
        time_t start, end;
        start = clock();
        trivialGemm<<<blocks, threads>>>(dA, dB, dC, m, n, k);
        hipError_t err = hipDeviceSynchronize();
        end = clock();
        printf("CUDA error: %s\n", hipGetErrorString(err));
        printf("Runtime of trivialGemm: %f\n", (double)(end - start) / CLOCKS_PER_SEC);
        hipMemcpy(C, dC, m * n * sizeof(double), hipMemcpyDeviceToHost);
        assertEqual(C, C_ref, m * n);
        printf("trivialGemm passed\n");
    }

    {
        // test trivialMMAWithoutCuTe
        randn(C, m * n, 0, 100);
        dim3 threads(BM * BN / 2);
        dim3 blocks(m / BM, n / BN);
        time_t start, end;
        start = clock();
        trivialMMAWithoutCuTe<BM, BN, BK><<<blocks, threads>>>(dA, dB, dC, m, n, k);
        hipError_t err = hipDeviceSynchronize();
        printf("CUDA error: %s\n", hipGetErrorString(err));
        end = clock();
        printf("Runtime of trivialMMAWithoutCuTe: %f\n", (double)(end - start) / CLOCKS_PER_SEC);
        hipMemcpy(C, dC, m * n * sizeof(double), hipMemcpyDeviceToHost);
        assertEqual(C, C_ref, m * n);
        printf("trivialMMAWithoutCuTe passed\n");
    }

    {
        // test trivialMMAWithCuTe
        randn(C, m * n, 0, 100);
        dim3 threads(BM * BN / 2);
        dim3 blocks(m / BM, n / BN);
        time_t start, end;
        start = clock();
        trivialMMAWithCuTe<BM, BN, BK><<<blocks, threads>>>(dA, dB, dC, m, n, k);
        hipError_t err = hipDeviceSynchronize();
        end = clock();
        printf("CUDA error: %s\n", hipGetErrorString(err));
        printf("Runtime of trivialMMAWithCuTe: %f\n", (double)(end - start) / CLOCKS_PER_SEC);
        hipMemcpy(C, dC, m * n * sizeof(double), hipMemcpyDeviceToHost);
        assertEqual(C, C_ref, m * n);
        printf("trivialMMAWithCuTe passed\n");
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);
    hipHostFree(C_ref);
    return 0;
}